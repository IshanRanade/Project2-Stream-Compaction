#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "common.h"
#include "naive.h"
#include <>

namespace StreamCompaction {
    namespace Naive {
        using StreamCompaction::Common::PerformanceTimer;
        PerformanceTimer& timer()
        {
            static PerformanceTimer timer;
            return timer;
        }

        // TODO: __global__
        __global__ void kernScan(int n, int power, int *read, int *write) {
            int index = threadIdx.x + (blockIdx.x * blockDim.x);

            if (index < n) {
                if (index >= power) {
                    write[index] = read[index - power] + read[index];
                }
                else {
                    write[index] = read[index];
                }
            }
        }
        /**
         * Performs prefix-sum (aka scan) on idata, storing the result into odata.
         */
        void scan(int n, int *odata, const int *idata) {
            // TODO

            int *dev_read;
            hipMalloc((void**)&dev_read, n * sizeof(int));

            int *dev_write;
            hipMalloc((void**)&dev_write, n * sizeof(int));

            hipMemcpy(dev_read, idata, n * sizeof(int), hipMemcpyHostToDevice);

            hipDeviceSynchronize();

            timer().startGpuTimer();

            int D = ilog2ceil(n);
            for (int d = 1; d < D + 1; ++d) {
                int blockSize = 256;
                int blocks = (n + blockSize - 1) / blockSize;

                int power = pow(2, d - 1);
                kernScan << <blocks, blockSize >> > (n, power, dev_read, dev_write);

                hipDeviceSynchronize();

                int *temp = dev_read;
                dev_read = dev_write;
                dev_write = temp;
            }

            timer().endGpuTimer();

            hipMemcpy(odata, dev_read, n * sizeof(int), hipMemcpyDeviceToHost);

            for (int i = n - 1; i >= 1; --i) {
                odata[i] = odata[i - 1];
            }
            odata[0] = 0;
           
        }

        
    }
}
