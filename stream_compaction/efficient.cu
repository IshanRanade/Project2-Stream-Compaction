#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "common.h"
#include "efficient.h"
#include <>
#include <iostream>
#include <stdlib.h>
#include <string>


namespace StreamCompaction {
    namespace Efficient {
        using StreamCompaction::Common::PerformanceTimer;
        PerformanceTimer& timer()
        {
            static PerformanceTimer timer;
            return timer;
        }

        __global__ void kernUpsweep(int n, int power, int *array) {
            int index = threadIdx.x + (blockIdx.x * blockDim.x);

            if (index < n) {
                if (index % (2 * power) == 0) {
                    array[index + (2 * power) - 1] = array[index + power - 1] + array[index + (2 * power) - 1];
                }
            }
        }

        __global__ void kernDownsweep(int n, int power, int *array) {
            int index = threadIdx.x + (blockIdx.x * blockDim.x);

            if (index < n) {
                if (index % (2 * power) == 0) {
                    int t = array[index + power - 1];
                    array[index + power - 1] = array[index + (2 * power) - 1];
                    array[index + (2 * power) - 1] = t + array[index + (2 * power) - 1];
                }
            }
        }

        __global__ void kernSetZero(int n, int *array) {
            int index = threadIdx.x + (blockIdx.x * blockDim.x);

            if (index < n) {
                if (index == n - 1) {
                    array[index] = 0;
                }
            }
        }

        /**
         * Performs prefix-sum (aka scan) on idata, storing the result into odata.
         */
        void scan(int n, int *odata, const int *idata) {
            int *temp;

            int size = 1;
            while (size < n) {
                size *= 2;
            }

            hipMalloc((void**)&temp, size * sizeof(int));
            hipDeviceSynchronize();

            hipMemcpy(temp, idata, n * sizeof(int), hipMemcpyHostToDevice);
            hipDeviceSynchronize();

            timer().startGpuTimer();

            int blockSize = 256;
            int blocks = (size + blockSize - 1) / blockSize;

            // TODO
            for (int d = 0; d < ilog2ceil(size); ++d) {
                kernUpsweep << <blocks, blockSize >> > (size, pow(2, d), temp);
                hipDeviceSynchronize();
            }
            
            kernSetZero << <blocks, blockSize >> > (size, temp);

            for (int d = ilog2ceil(size) - 1; d >= 0; --d) {
                kernDownsweep << <blocks, blockSize >> > (size, pow(2, d), temp);
                hipDeviceSynchronize();
            }

            timer().endGpuTimer();

            hipMemcpy(odata, temp, n * sizeof(int), hipMemcpyDeviceToHost);
        }

        __global__ void kernMapToBoolean(int n, int *read, int *write) {
            int index = threadIdx.x + (blockIdx.x * blockDim.x);

            if (index < n) {
                if (read[index] == 0) {
                    write[index] = 0;
                }
                else {
                    write[index] = 1;
                }
            }
        }

        __global__ void kernScatter(int n, int *idata, int *booleans, int *scan, int *odata) {
            int index = threadIdx.x + (blockIdx.x * blockDim.x);

            if (index < n) {
                if (booleans[index] == 1) {
                    odata[scan[index]] = idata[index];
                }
            }
        }

        /**
         * Performs stream compaction on idata, storing the result into odata.
         * All zeroes are discarded.
         *
         * @param n      The number of elements in idata.
         * @param odata  The array into which to store elements.
         * @param idata  The array of elements to compact.
         * @returns      The number of elements remaining after compaction.
         */
        int compact(int n, int *odata, const int *idata) {
            //timer().startGpuTimer();

            int blockSize = 256;
            int blocks = (n + blockSize - 1) / blockSize;

            int *dev_idata;
            int *booleans;
            int *scanArray;
            int *result;

            hipMallocManaged(&dev_idata, n * sizeof(int));

            hipDeviceSynchronize();
            
            for (int i = 0; i < n; ++i) {
                dev_idata[i] = idata[i];
            }
            
            hipMallocManaged(&booleans, n * sizeof(int));
            hipMallocManaged(&scanArray, n * sizeof(int));
            hipMallocManaged(&result, n * sizeof(int));

            hipDeviceSynchronize();

            // First map the initial array to booleans
            kernMapToBoolean << <blocks, blockSize >> > (n, dev_idata, booleans);

            hipDeviceSynchronize();

            // Now do a scan
            scan(n, scanArray, booleans);

            // Now do a scatter
            int *dev_odata;
            hipMallocManaged(&dev_odata, n * sizeof(int));
            kernScatter << <blocks, blockSize >> > (n, dev_idata, booleans, scanArray, dev_odata);

            hipDeviceSynchronize();

            int finalCount = 0;
            for (int i = 0; i < n; ++i) {
                finalCount += booleans[i];
            }

            hipMemcpy(odata, dev_odata, finalCount * sizeof(int), hipMemcpyDeviceToHost);

            return finalCount;
        }
    }
}
